#include "hip/hip_runtime.h"
//para compilar:
//make
//para ejecutar
//./wave [-N numero entero] [-x numero entero] [-y numero entero] [-T numero entero] [-f cadena caracteres]
//./wave -N 30 -x 5 -y 5 -T 1 -f imagen.raw
extern "C" {
  

#include <stdio.h>
#include <stdlib.h>
#include "funciones.h"
}

__global__ void wave(float *H1, float *H2, float*HAUX, int contador, int N)
{
  float c=1.0 , dt=0.1 , dd=2.0;
  float iMenos1,iMas1,jMenos1,jMas1;
  int j, i;
  i = blockDim.x*blockIdx.x + threadIdx.x;  // global index x (horizontal)
  j = blockDim.y*blockIdx.y + threadIdx.y;  // global index y (vertical)
  
  //caso inicial
  if( contador ==1)
  {
    iMenos1=HAUX[(i-1)*N+j];
    jMenos1=HAUX[i*N+(j-1)];
    iMas1=HAUX[(i+1)*N+j];
    jMas1=HAUX[i*N+(j+1)];

    H1[i*N+j]= HAUX[i*N+j]+ (c*c)*((dt/dd)*(dt/dd))*(iMas1+iMenos1+jMenos1+jMas1-4*HAUX[i*N+j]); 
  }
  else //caso normal
  {     
    iMenos1=HAUX[(i-1)*N+j];
    jMenos1=HAUX[i*N+(j-1)];
    iMas1=HAUX[(i+1)*N+j];
    jMas1=HAUX[i*N+(j+1)];

    H1[i*N+j]= 2*HAUX[i*N+j]-H2[i*N+j] +(c*c)*((dt/dd)*(dt/dd))*(iMas1+iMenos1+jMenos1+jMas1-4*HAUX[i*N+j]);

  }

  return H1;
    
}



//Main

__host__ int main(int argc, char *argv[])
{
/////////////////////////////////////////////////////
//  INICIO:   Lectura de argumentos
/////////////////////////////////////////////////////


  int N=0,T=0, x=0, y=0;
  char * f = NULL;
  f=recibirArgumentos(argc, argv, &N, &T, f, &x, &y);

/////////////////////////////////////////////////////
//  FIN:    Lectura de argumentos
///////////////////////////////////////////////////// 


  
/////////////////////////////////////////////////////
//  INICIO:   LLenado inicial de la matriz de host
/////////////////////////////////////////////////////
  
  float *H1 = (float *) malloc(N*N*sizeof(float));
  float *H2 = (float *) malloc(N*N*sizeof(float));
  float *HAUX = (float *) malloc(N*N*sizeof(float));
  

  for (int i = 0; i < N; i++)
  {
    for (int j = 0; j < N; j++)
    {
      if( 0.4*N < i && 0.6*N > i && 0.4*N < j && 0.6*N > j) 
      {     
        H1[i*N+j]=20;
        H2[i*N+j]=20;
        HAUX[i*N+j]=20;
      }
      else
      {
        H1[i*N+j]=0;
        H2[i*N+j]=0;
        HAUX[i*N+j]=0;
      }
    }
  }        
  
/////////////////////////////////////////////////////
//  FIN:   LLenado inicial de la matriz host
///////////////////////////////////////////////////// 

/////////////////////////////////////////////////////
//  INICIO:   Asignacion de memoria Device
/////////////////////////////////////////////////////

  float *d_H1;
  float *d_H2;
  float *d_HAUX;

  //Se asigna memoria para cada vector del device
  hipMalloc( &d_H1, N*N*sizeof(float));
  hipMalloc( &d_H2, N*N*sizeof(float));
  hipMalloc( &d_HAUX, N*N*sizeof(float));

  //Se copia el vector del host al vector del device
  hipMemcpy(d_H1, H1, N*N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_H2, H2, N*N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_HAUX, HAUX, N*N*sizeof(float), hipMemcpyHostToDevice);

  /////////////////////////////////////////////////////
//  FIN:   Asignacion de memoria Device
/////////////////////////////////////////////////////


/////////////////////////////////////////////////////
//  INICIO:   Calculo de ecuacion
///////////////////////////////////////////////////// 

  int contador = 1;

  dim3 blocksize;
  dim3 gridsize;

  //Numero de threads en cada bloque

  gridsize.x = N/x;
  gridsize.y = N/y;
  blocksize.x = x;
  blocksize.y = y;



  //Se ejecuta el kernel
  

  while( contador <= T)
  {
    swap(H1,HAUX,N);
    hipMemcpy(d_H1, H1, N*N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_H2, H2, N*N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_HAUX, HAUX, N*N*sizeof(float), hipMemcpyHostToDevice);
    wave<<<gridsize, blocksize>>>(d_H1, d_H2, d_HAUX, contador, N);    
    hipMemcpy(d_H1, H1, N*N*sizeof(double), hipMemcpyDeviceToHost);
    swap(HAUX,H2,N);    
    contador++;
    
    
  }      

/////////////////////////////////////////////////////
//  FIN:   Calculo de ecuacion
///////////////////////////////////////////////////// 

  

  FILE *f1 = fopen(f, "w");
  fwrite(H1, sizeof(float), N*N, f1);
  fclose(f1);

  hipFree(d_H1);
  hipFree(d_H2);
  hipFree(d_HAUX);

  free(H1);
  free(H2);
  free(HAUX);

  return 0;
}