#include "hip/hip_runtime.h"
//para compilar:
//make
//para ejecutar
//./wave [-N numero entero] [-x numero entero] [-y numero entero] [-T numero entero] [-f cadena caracteres]
//./wave -N 30 -x 5 -y 5 -T 1 -f imagen.raw
extern "C" {
  

#include <stdio.h>
#include <stdlib.h>
#include "funciones.h"
}

__global__ void wave(float *H1, float *H2, float*HAUX, int contador)
{

  int blocksize = blockDim.y * blockDim.x; // number of threads in a TB
  int blockId = gridDim.x * blockIdx.y + blockIdx.x; // unique block Id
  int tid = blockId * blocksize + // number of threads before this block
  blockDim.x*threadIdx.y + threadIdx.x; // global tid
  printf("I am thread (%d, %d) in block (%d, %d). Global thread ID = %d\n",
  threadIdx.y, threadIdx.x, blockIdx.y, blockIdx.x, tid);
    /*obtencion del id global
    int i, j;
    i = blockDim.x*blockIdx.x + threadIdx.x;  // global index x (horizontal)
    j = blockDim.y*blockIdx.y + threadIdx.y;  // global index y (vertical)

    printf("El i: %d   el j: %d  el blockX: %d el blockY: %d \n",i,j,blockDim.x,blockDim.y);

    
    //caso inicial
    if( contador ==1)
    {
    
      for (int i = 1; i < N-1; i++)
      {        
        for (int j = 1; j < N-1; j++)
        {
          iMenos1=HAUX[(i-1)*N+j];
          jMenos1=HAUX[i*N+(j-1)];
          iMas1=HAUX[(i+1)*N+j];
          jMas1=HAUX[i*N+(j+1)];

          H1[i*N+j]= HAUX[i*N+j]+ (c*c)*((dt/dd)*(dt/dd))*(iMas1+iMenos1+jMenos1+jMas1-4*HAUX[i*N+j]);

        } 

      }  
    }
    else //caso normal
    {
      for (int i = 1; i < N-1; i++)
      {        
        for (int j = 1; j < N-1; j++)
        {        
          iMenos1=HAUX[(i-1)*N+j];
          jMenos1=HAUX[i*N+(j-1)];
          iMas1=HAUX[(i+1)*N+j];
          jMas1=HAUX[i*N+(j+1)];

          H1[i*N+j]= 2*HAUX[i*N+j]-H2[i*N+j] +(c*c)*((dt/dd)*(dt/dd))*(iMas1+iMenos1+jMenos1+jMas1-4*HAUX[i*N+j]);
        }
        
      }

    }
    */
}



//Main

__host__ int main(int argc, char *argv[])
{
/////////////////////////////////////////////////////
//  INICIO:   Lectura de argumentos
/////////////////////////////////////////////////////


  int N=0,T=0, x=0, y=0;
  char * f = NULL;
  f=recibirArgumentos(argc, argv, &N, &T, f, &x, &y);

/////////////////////////////////////////////////////
//  FIN:    Lectura de argumentos
///////////////////////////////////////////////////// 


  
/////////////////////////////////////////////////////
//  INICIO:   LLenado inicial de la matriz de host
/////////////////////////////////////////////////////
  
  float *H1 = (float *) malloc(N*N*sizeof(float));
  float *H2 = (float *) malloc(N*N*sizeof(float));
  float *HAUX = (float *) malloc(N*N*sizeof(float));
  

  for (int i = 0; i < N; i++)
  {
    for (int j = 0; j < N; j++)
    {
      if( 0.4*N < i && 0.6*N > i && 0.4*N < j && 0.6*N > j) 
      {     
        H1[i*N+j]=20;
        H2[i*N+j]=20;
        HAUX[i*N+j]=20;
      }
      else
      {
        H1[i*N+j]=0;
        H2[i*N+j]=0;
        HAUX[i*N+j]=0;
      }
    }
  }        
  
/////////////////////////////////////////////////////
//  FIN:   LLenado inicial de la matriz host
///////////////////////////////////////////////////// 

/////////////////////////////////////////////////////
//  INICIO:   Asignacion de memoria Device
/////////////////////////////////////////////////////

  float *d_H1;
  float *d_H2;
  float *d_HAUX;

  //Se asigna memoria para cada vector del device
  hipMalloc( &d_H1, N*N*sizeof(float));
  hipMalloc( &d_H2, N*N*sizeof(float));
  hipMalloc( &d_HAUX, N*N*sizeof(float));

  //Se copia el vector del host al vector del device
  hipMemcpy(d_H1, H1, N*N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_H2, H2, N*N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_HAUX, HAUX, N*N*sizeof(float), hipMemcpyHostToDevice);

  /////////////////////////////////////////////////////
//  FIN:   Asignacion de memoria Device
/////////////////////////////////////////////////////


/////////////////////////////////////////////////////
//  INICIO:   Calculo de ecuacion
///////////////////////////////////////////////////// 

/*
  float c=1.0 , dt=0.1 , dd=2.0;
  float iMenos1,iMas1,jMenos1,jMas1;
  */
  int contador = 1;

  dim3 blocksize;
  dim3 gridsize;

  //Numero de threads en cada bloque

  gridsize.x = N;
  gridsize.y = N;
  blocksize.x = x;
  blocksize.y = y;



  //Se ejecuta el kernel
  

  while( contador <= T)
  {
    swap(H1,HAUX,N);
    wave<<<gridsize, blocksize>>>(d_H1, d_H2, d_HAUX, contador);
    swap(HAUX,H2,N);    
    contador++;
    
    
  }      

/////////////////////////////////////////////////////
//  FIN:   Calculo de ecuacion
///////////////////////////////////////////////////// 

  

  FILE *f1 = fopen(f, "w");
  fwrite(H1, sizeof(float), N*N, f1);
  fclose(f1);

  hipFree(d_H1);
  hipFree(d_H2);
  hipFree(d_HAUX);

  free(H1);
  free(H2);
  free(HAUX);

  return 0;
}