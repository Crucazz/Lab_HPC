#include "hip/hip_runtime.h"
//para compilar:
//make
//para ejecutar
//./wave [-N numero entero] [-x numero entero] [-y numero entero] [-T numero entero] [-f cadena caracteres]
//./wave -N 30 -x 5 -y 5 -T 1 -f imagen.raw
extern "C" {
  

#include <stdio.h>
#include <stdlib.h>
#include "funciones.h"
}

__global__ void wave(float *H1, float *H2, float*HAUX, int N, int contador)
{
  float c=1.0 , dt=0.1 , dd=2.0;
  float iMenos1,iMas1,jMenos1,jMas1;
  int j, i;
  extern __shared__ float temp[];

  i = blockDim.x*blockIdx.x + threadIdx.x;  // global index x (horizontal)
  j = blockDim.y*blockIdx.y + threadIdx.y;  // global index y (vertical)
    //caso inicial
  if( contador ==1)
  {
    iMenos1=HAUX[(i-1)*N+j];
    jMenos1=HAUX[i*N+(j-1)];
    iMas1=HAUX[(i+1)*N+j];
    jMas1=HAUX[i*N+(j+1)];

    temp[i*N+j]= HAUX[i*N+j]+ (c*c)*((dt/dd)*(dt/dd))*(iMas1+iMenos1+jMenos1+jMas1-4*HAUX[i*N+j]); 
  }
  else //caso normal
  {     
    iMenos1=HAUX[(i-1)*N+j];
    jMenos1=HAUX[i*N+(j-1)];
    iMas1=HAUX[(i+1)*N+j];
    jMas1=HAUX[i*N+(j+1)];

    temp[i*N+j]= 2*HAUX[i*N+j]-H2[i*N+j] +(c*c)*((dt/dd)*(dt/dd))*(iMas1+iMenos1+jMenos1+jMas1-4*HAUX[i*N+j]);
    printf("Temp = %f \n",temp[i*N+j]);
  }
  __syncthreads(); 
  if (i == 1 && j==1) {    
    printf("Se procede a actualizar H1\n");
    for (i = 1; i < N-1; i++)
      {        
        for (j = 1; j < N-1; j++){
          H1[i*N+j] =temp[i*N+j];        
        }
      }
    
  }
    
}

__global__ void swap2(float *origen,float *destino, int N)
{
  int j, i;
  i = blockDim.x*blockIdx.x + threadIdx.x;  // global index x (horizontal)
  j = blockDim.y*blockIdx.y + threadIdx.y;  // global index y (vertical)
  destino[i*N+j]=origen[i*N+j];
  
}



//Main

__host__ int main(int argc, char *argv[])
{
/////////////////////////////////////////////////////
//  INICIO:   Lectura de argumentos
/////////////////////////////////////////////////////


  int N=0,T=0, x=0, y=0;
  char * f = NULL;
  f=recibirArgumentos(argc, argv, &N, &T, f, &x, &y);

/////////////////////////////////////////////////////
//  FIN:    Lectura de argumentos
///////////////////////////////////////////////////// 


  
/////////////////////////////////////////////////////
//  INICIO:   LLenado inicial de la matriz de host
/////////////////////////////////////////////////////
  
  float *H1 = (float *) malloc(N*N*sizeof(float));
  float *H2 = (float *) malloc(N*N*sizeof(float));
  float *HAUX = (float *) malloc(N*N*sizeof(float));
  
  int flag = 0;
  for (int i = 0; i < N; i++)
  {
    for (int j = 0; j < N; j++)
    {
      if( 0.4*N < i && 0.6*N > i && 0.4*N < j && 0.6*N > j) 
      {     
        if(flag==0){
          printf("(%d,%d)=20\n",i,j);
          flag=1;
        }
        
        H1[i*N+j]=20;
        H2[i*N+j]=20;
        HAUX[i*N+j]=20;
      }
      else
      {
        H1[i*N+j]=0;
        H2[i*N+j]=0;
        HAUX[i*N+j]=0;
      }
    }
  }        
  
/////////////////////////////////////////////////////
//  FIN:   LLenado inicial de la matriz host
///////////////////////////////////////////////////// 

/////////////////////////////////////////////////////
//  INICIO:   Asignacion de memoria Device
/////////////////////////////////////////////////////

  float *d_H1;
  float *d_H2;
  float *d_HAUX;

  //Se asigna memoria para cada vector del device
  hipMalloc( &d_H1, N*N*sizeof(float));
  hipMalloc( &d_H2, N*N*sizeof(float));
  hipMalloc( &d_HAUX, N*N*sizeof(float));

  //Se copia el vector del host al vector del device
  hipMemcpy(d_H1, H1, N*N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_H2, H2, N*N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_HAUX, HAUX, N*N*sizeof(float), hipMemcpyHostToDevice);

  /////////////////////////////////////////////////////
//  FIN:   Asignacion de memoria Device
/////////////////////////////////////////////////////


/////////////////////////////////////////////////////
//  INICIO:   Calculo de ecuacion
///////////////////////////////////////////////////// 

  dim3 blocksize;
  dim3 gridsize;

  //Numero de threads en cada bloque

  gridsize.x = N/x;
  gridsize.y = N/y;
  blocksize.x = x;
  blocksize.y = y;

  int contador = 1;


  //Se ejecuta el kernel
  while( contador <= T)
  {
    swap2<<<gridsize, blocksize>>>(d_H1,d_HAUX,N);
    hipDeviceSynchronize();
    wave<<<gridsize, blocksize, N*N*sizeof(float)>>>(d_H1, d_H2, d_HAUX, N, contador);
    hipError_t err = hipGetLastError();
            if (err != hipSuccess) 
              printf("Error: %s\n", hipGetErrorString(err));
    swap2<<<gridsize, blocksize>>>(d_HAUX,d_H2,N);    
    hipDeviceSynchronize();
    contador++;
    
  }      
  printf("TErmino");
    
    
  

/////////////////////////////////////////////////////
//  FIN:   Calculo de ecuacion
///////////////////////////////////////////////////// 
  hipMemcpy(H1, d_H1, N*N*sizeof(float), hipMemcpyDeviceToHost);
  FILE *f1 = fopen(f, "w");
  fwrite(H1, sizeof(float), N*N, f1);
  fclose(f1);

  hipFree(d_H1);
  hipFree(d_H2);
  hipFree(d_HAUX);

  free(H1);
  free(H2);
  free(HAUX);

  return 0;
}