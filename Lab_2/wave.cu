#include "hip/hip_runtime.h"
//para compilar:
//make
//para ejecutar
//./wave [-N numero entero] [-x numero entero] [-y numero entero] [-T numero entero] [-f cadena caracteres]
//./wave -N 30 -x 5 -y 5 -T 1 -f imagen.raw
extern "C" {
  

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "funciones.h"
}

__global__ void wave(float *H1, float *H2, float*HAUX, int N, int contador)
{
  float c=1.0 , dt=0.1 , dd=2.0;
  float iMenos1,iMas1,jMenos1,jMas1;
  int j, i;
  extern __shared__ float temp[];

  i = blockDim.x*blockIdx.x + threadIdx.x;  // global index x (horizontal)
  j = blockDim.y*blockIdx.y + threadIdx.y;  // global index y (vertical)
    //caso inicial
  if (i >1 && i<N-1 && j>1 && j<N-1 )
  {  
    if( contador ==1)
    {
      iMenos1=HAUX[(i-1)*N+j];
      jMenos1=HAUX[i*N+(j-1)];
      iMas1=HAUX[(i+1)*N+j];
      jMas1=HAUX[i*N+(j+1)];

      temp[(threadIdx.x*blockDim.x)+threadIdx.y]= HAUX[i*N+j]+ (c*c)*((dt/dd)*(dt/dd))*(iMas1+iMenos1+jMenos1+jMas1-4*HAUX[i*N+j]);
      
    }
    else //caso normal
    {
      iMenos1=HAUX[(i-1)*N+j];
      jMenos1=HAUX[i*N+(j-1)];
      iMas1=HAUX[(i+1)*N+j];
      jMas1=HAUX[i*N+(j+1)];
      temp[(threadIdx.x*blockDim.x)+threadIdx.y]= 2*HAUX[i*N+j]-H2[i*N+j] +(c*c)*((dt/dd)*(dt/dd))*(iMas1+iMenos1+jMenos1+jMas1-4*HAUX[i*N+j]);
     
    }
    
    H1[i*N+j] =temp[(threadIdx.x*blockDim.x)+threadIdx.y];
  }
    
  

}

__global__ void swap(float *origen,float *destino, int N)
{
  int j, i;
  i = blockDim.x*blockIdx.x + threadIdx.x;  // global index x (horizontal)
  j = blockDim.y*blockIdx.y + threadIdx.y;  // global index y (vertical)
  if (i >1 && i<N-1 && j>1 && j<N-1 )
    destino[i*N+j]=origen[i*N+j];
  
}



//Main

__host__ int main(int argc, char *argv[])
{
/////////////////////////////////////////////////////
//  INICIO:   Lectura de argumentos
/////////////////////////////////////////////////////


  int N=0,T=0, x=0, y=0;
  char * f = NULL;
  f=recibirArgumentos(argc, argv, &N, &T, f, &x, &y);

/////////////////////////////////////////////////////
//  FIN:    Lectura de argumentos
///////////////////////////////////////////////////// 


  
/////////////////////////////////////////////////////
//  INICIO:   LLenado inicial de la matriz de host
/////////////////////////////////////////////////////
  
  float *H1 = (float *) malloc(N*N*sizeof(float));
  float *H2 = (float *) malloc(N*N*sizeof(float));
  float *HAUX = (float *) malloc(N*N*sizeof(float));

  for (int i = 0; i < N; i++)
  {
    for (int j = 0; j < N; j++)
    {
      if( 0.4*N < i && 0.6*N > i && 0.4*N < j && 0.6*N > j) 
      {             
        H1[i*N+j]=20;
        H2[i*N+j]=20;
        HAUX[i*N+j]=20;
      }
      else
      {
        H1[i*N+j]=0;
        H2[i*N+j]=0;
        HAUX[i*N+j]=0;
      }
    }
  }        
  
/////////////////////////////////////////////////////
//  FIN:   LLenado inicial de la matriz host
///////////////////////////////////////////////////// 

/////////////////////////////////////////////////////
//  INICIO:   Asignacion de memoria Device
/////////////////////////////////////////////////////

  float *d_H1;
  float *d_H2;
  float *d_HAUX;

  //Se asigna memoria para cada vector del device
  hipMalloc( &d_H1, N*N*sizeof(float));
  hipMalloc( &d_H2, N*N*sizeof(float));
  hipMalloc( &d_HAUX, N*N*sizeof(float));

  //Se copia el vector del host al vector del device
  hipMemcpy(d_H1, H1, N*N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_H2, H2, N*N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_HAUX, HAUX, N*N*sizeof(float), hipMemcpyHostToDevice);

  /////////////////////////////////////////////////////
//  FIN:   Asignacion de memoria Device
/////////////////////////////////////////////////////


/////////////////////////////////////////////////////
//  INICIO:   Calculo de ecuacion
///////////////////////////////////////////////////// 

  dim3 blocksize;
  dim3 gridsize;

  //Numero de threads en cada bloque

  int BiasX = 0, BiasY = 0;

  if (N%x!=0)
    BiasX = 1;
  if (N%y!=0)
    BiasY = 1;
  gridsize.x = (N/x)+BiasX;
  gridsize.y = (N/y)+BiasY;
  blocksize.x = x;
  blocksize.y = y;

  //printf("El tamaño de N: %d, gridZise: %d * %d\n",N,gridsize.x,gridsize.y);
  int contador = 1;

  int numBlocks = gridsize.x*gridsize.y;
  int TamBloque = x*y;
  int device;
  hipDeviceProp_t prop;
  int activeWarps;
  int maxWarps;

  hipGetDevice(&device);
  hipGetDeviceProperties(&prop, device);  
  hipOccupancyMaxActiveBlocksPerMultiprocessor( &numBlocks,
        wave,
        TamBloque,
        0);

    activeWarps = numBlocks * TamBloque / prop.warpSize;
    maxWarps = prop.maxThreadsPerMultiProcessor / prop.warpSize;

    printf ("La ocupancia %lf  active/max*100 \n",(double)activeWarps / maxWarps * 100);
    printf ("MAx warp: %d   y  activos warp : %d \n",maxWarps,activeWarps);




  hipEvent_t start2, stop2;
  float gpu_time = 0.0f;
  hipEventCreate(&start2) ;
  hipEventCreate(&stop2) ;
  hipEventRecord(start2, 0);  

  time_t start = time(NULL);

  //Se ejecuta el kernel
  while( contador <= T)
  {
    swap<<<gridsize, blocksize>>>(d_H1,d_HAUX,N);
    hipDeviceSynchronize();
    wave<<<gridsize, blocksize, x*y*sizeof(float)>>>(d_H1, d_H2, d_HAUX, N, contador);
    hipError_t err = hipGetLastError();
            if (err != hipSuccess)
              printf("Error: %s\n", hipGetErrorString(err));
    swap<<<gridsize, blocksize>>>(d_HAUX,d_H2,N);
    hipDeviceSynchronize();
    contador++;

  }
  time_t finish = time(NULL);
  hipEventRecord(stop2, 0);
  hipEventSynchronize(stop2);
  hipEventElapsedTime(&gpu_time, start2, stop2);
  printf("Time spent: %.5f\n", gpu_time);
  hipEventDestroy(start2);
  hipEventDestroy(stop2);
  printf("Wall-Clock: %.5f\n", (double)(finish - start));




/////////////////////////////////////////////////////
//  FIN:   Calculo de ecuacion
/////////////////////////////////////////////////////
  hipMemcpy(H1, d_H1, N*N*sizeof(float), hipMemcpyDeviceToHost);
  FILE *f1 = fopen(f, "w");
  fwrite(H1, sizeof(float), N*N, f1);
  fclose(f1);

  hipFree(d_H1);
  hipFree(d_H2);
  hipFree(d_HAUX);

  free(H1);
  free(H2);
  free(HAUX);

  return 0;
}